#include <cstdio>
#include "merge_kernel.cuh"

float* circular_merge(float *a, float *b, int n, int m) {
    float* c = (float*) malloc(sizeof(float) * (n + m));
    float *a_d, *b_d, *c_d;
    hipMalloc((void**) &a_d, sizeof(float) * n);
    hipMalloc((void**) &b_d, sizeof(float) * m);
    hipMalloc((void**) &c_d, sizeof(float) * (n + m));
    hipMemcpy(a_d, a, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(float) * m, hipMemcpyHostToDevice);

    dim3 gridDim((n + m + TILE_SIZE * COARSE_FACTOR - 1) / (TILE_SIZE * COARSE_FACTOR));
    dim3 blockDim(TILE_SIZE);

    circularMergeSort<<<gridDim, blockDim>>>(a_d, b_d, c_d, n, m, n + m);
    hipDeviceSynchronize();
    hipMemcpy(c, c_d, sizeof(float) * (n + m), hipMemcpyDeviceToHost);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return c;
}